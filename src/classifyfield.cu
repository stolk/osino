#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_fp16.h>

#define BLKRES	(1<<BLKMAG)
#define BLKMSK	(BLKRES-1)

#if defined(STOREFP16)
typedef __half value_t;
#elif defined(STORECHARS)
typedef unsigned char value_t;
#elif defined(STORESHORTS)
typedef short value_t;
#else
typedef float value_t;
#endif


extern "C"
{
__global__
void osino_classifyfield
(
	value_t isoval,
	const value_t* field,
	uint8_t* cases
)
{
	const int zc = threadIdx.x;
	const int yc = blockIdx.x & BLKMSK;
	const int xc = (blockIdx.x >> BLKMAG);

	const int i0 = zc + yc*BLKRES + xc*BLKRES*BLKRES;
	//assert(i0 < BLKRES*BLKRES*BLKRES);

	const int stridex = xc < BLKRES-1 ? BLKRES*BLKRES : 0;
	const int stridey = yc < BLKRES-1 ? BLKRES : 0;
	const int stridez = zc < BLKRES-1 ? 1 : 0;

	const int i1 = i0 + stridex;
	const int i2 = i1 + stridey;
	const int i3 = i0 + stridey;

	const int i4 = i0+stridez;
	const int i5 = i1+stridez;
	const int i6 = i2+stridez;
	const int i7 = i3+stridez;

#if defined(STORECHARS)
	const float scl = 1.0f / 128.0f
	const float v0 = -1.0f + field[i0]*scl;
	const float v1 = -1.0f + field[i1]*scl;
	const float v2 = -1.0f + field[i2]*scl;
	const float v3 = -1.0f + field[i3]*scl;
	const float v4 = -1.0f + field[i4]*scl;
	const float v5 = -1.0f + field[i5]*scl;
	const float v6 = -1.0f + field[i6]*scl;
	const float v7 = -1.0f + field[i7]*scl;
#elif defined(STORESHORTS)
	const value_t v0 = field[i0];
	const value_t v1 = field[i1];
	const value_t v2 = field[i2];
	const value_t v3 = field[i3];
	const value_t v4 = field[i4];
	const value_t v5 = field[i5];
	const value_t v6 = field[i6];
	const value_t v7 = field[i7];
#elif defined(STOREFP16)
	const float v0 = __half2float(field[i0]);
	const float v1 = __half2float(field[i1]);
	const float v2 = __half2float(field[i2]);
	const float v3 = __half2float(field[i3]);
	const float v4 = __half2float(field[i4]);
	const float v5 = __half2float(field[i5]);
	const float v6 = __half2float(field[i6]);
	const float v7 = __half2float(field[i7]);
#else
	const float v0 = field[i0];
	const float v1 = field[i1];
	const float v2 = field[i2];
	const float v3 = field[i3];
	const float v4 = field[i4];
	const float v5 = field[i5];
	const float v6 = field[i6];
	const float v7 = field[i7];
#endif
	const int bit0 = v0 <= isoval ? 0x01 : 0;
	const int bit1 = v1 <= isoval ? 0x02 : 0;
	const int bit2 = v2 <= isoval ? 0x04 : 0;
	const int bit3 = v3 <= isoval ? 0x08 : 0;
	const int bit4 = v4 <= isoval ? 0x10 : 0;
	const int bit5 = v5 <= isoval ? 0x20 : 0;
	const int bit6 = v6 <= isoval ? 0x40 : 0;
	const int bit7 = v7 <= isoval ? 0x80 : 0;

	const uint8_t c = bit0|bit1|bit2|bit3|bit4|bit5|bit6|bit7;
	cases[i0] = c;
}


__global__
void osino_setupfield(value_t* field)
{
	const int zc = threadIdx.x;
	const int yc = blockIdx.x & BLKMSK;
	const int xc = (blockIdx.x >> BLKMAG);

	const int i0 = zc + yc*BLKRES + xc*BLKRES*BLKRES;

	float x = 0.5f * BLKRES - xc;
	float y = 0.5f * BLKRES - yc;
	float z = 0.5f * BLKRES - zc;
	const float scl = 2.0f / BLKRES;
	float d = sqrtf(x*x + y*y + z*z) * scl;
#if defined(STOREFP16)
	field[i0] = __float2half(d);
#elif defined(STORESHORTS)
	d = d < -1 ? -1 : d;
	d = d >  1 ?  1 : d;
	field[i0] = (value_t) (d * 32767.0f);
#endif
}


__host__
void query(void)
{
	int nDevices=-1;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		int maxthr=-1;
		hipDeviceGetAttribute(&maxthr, hipDeviceAttributeMaxThreadsPerBlock, i);
		int wrpsiz=-1;
		hipDeviceGetAttribute(&wrpsiz, hipDeviceAttributeWarpSize, i);
		fprintf(stderr, "Device Number: %d\n", i);
		fprintf(stderr, "  Device name: %s\n", prop.name);
		fprintf(stderr, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		fprintf(stderr, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		fprintf(stderr, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		fprintf(stderr, "  Max threads per block: %d\n", maxthr);
		fprintf(stderr, "  Warp size: %d\n", wrpsiz);
	}
}

#define CHECK_CUDA \
	{ \
		const hipError_t err = hipGetLastError(); \
		fprintf(stderr,"%s\n", hipGetErrorString(err)); \
	}
}


__host__
int main(int argc, char* argv[])
{
	query();

	const int N = BLKRES*BLKRES*BLKRES;

	value_t* field = 0;
	hipMallocManaged(&field, N*sizeof(value_t));
	assert(field);
	CHECK_CUDA

	uint8_t* cases = 0;
	hipMallocManaged(&cases, N*sizeof(uint8_t));
	assert(cases);
	CHECK_CUDA

	osino_setupfield<<<BLKRES*BLKRES,BLKRES>>>( field );
	CHECK_CUDA

	osino_classifyfield<<<BLKRES*BLKRES,BLKRES>>>( 28000, field, cases );
	CHECK_CUDA

	hipDeviceSynchronize();
	CHECK_CUDA

	FILE* f = fopen("out_classify.pgm","wb");
	fprintf(f, "P5\n%d %d\n255\n", BLKRES, BLKRES);
	const uint8_t* reader = cases + (BLKRES/2)*BLKRES*BLKRES;
	for (int i=0; i<BLKRES*BLKRES; ++i)
		fputc(reader[i],f);
	fclose(f);

	hipFree(cases);
	hipFree(field);

	return 0;
}

